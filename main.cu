#include "hip/hip_runtime.h"
#ifndef INTERACTIONS_H
#define INTERACTIONS_H
#define GL_H
#define GL_GLEXT_PROTOTYPES
#define GRAPHICS_H
#endif

#define cimg_display 0

#include <cstdio>
#include <cmath>
#include <cstdlib>
#include <GL/gl.h>
#include <GL/freeglut.h>
#include <hip/hip_runtime.h>
#include <cudaGL.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_texture_types.h>
#include <>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <CImg.h>




using namespace std;
using namespace cimg_library;

#define REFRESH_DELAY 10 
#define DELTA 10
#define W 512
#define H 512

#define TX 16
#define TY 16


//Graphics Resource objects
GLuint pbo = 0;
GLuint tex = 0;
struct hipGraphicsResource *cuda_pbo_resource;
struct uchar4;
struct float4;
struct int3 loc = {W/2, H/2, 1};

// Parameters initialization
int sys = 2;
float param = 0.25f;
GLfloat angle1 = 0.0f;
GLfloat angle2 = 0.0f;
float g_fAnim = 0.0;
int mouse_old_x; 
int mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0;
float rotate_y = 0.0;
float translate_z = -3.0;
float scale = 500;




// Cuda kernel model
__device__
unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

__device__
int idxClip(int idx, int idxMax) {
return idx > (idxMax-1) ? (idxMax-1) : (idx < 0 ? 0 : idx);
}

__device__
int flatten(int col, int row, int width, int height) {
return idxClip(col, width) + idxClip(row, height)*width;
}


__global__
void distanceKernel(uchar4 *d_out, float4  *d_in, int w, int h, int3 pos, float time) {
    const int c = blockIdx.x*blockDim.x + threadIdx.x;
    const int r = blockIdx.y*blockDim.y + threadIdx.y;
    
    if ((c >= w) || (r >= h)) return;
    
    const int i = flatten(c, r, w, h);
    
    float u = c / (float) w;
    float v = r / (float) h;
    
    u = u*4.0f - 1.0f;
    v = v*4.0f - 1.0f;
    
    
    float freq, radius;
    freq= 2.25;
    radius = 10.0f;
    
    float a = (u*freq + time);
    float b = (v*freq + time);   

    
    //Equation   
    float dist1 = time*( 1-radius*sinf(a) * cosf(b));         
    float dist2 = sqrtf((c - pos.x)*(c - pos.x)*d_in[i].x + (r - pos.y)*(r - pos.y)* d_in[i].y - pos.z* d_in[i].z);

    const unsigned char intensity1 = clip(255 - dist1*d_in[i].x);
    const unsigned char intensity2 = clip(255 - dist2*d_in[i].y);
    const unsigned char intensity3 = clip(255 - dist1*dist2*d_in[i].z);

    
    d_out[i].x = intensity1;
    d_out[i].y = intensity3;
    d_out[i].z = intensity2;
    d_out[i].w = 255;
}


// Cuda run model
static void cudarun() {
    
    CImg<float> img (W,H,1,3);
    img.fill(200);
    
    //input data
    float4 *arr=(float4*)malloc(W*H*sizeof(float4));
    
    // Copy data to array
    for (int r = 0; r < H; ++r) {
        for (int c = 0; c < W; ++c){
            arr[r*W + c].x = img(c,r, 0);
            arr[r*W + c].y = img(c,r, 0);
            arr[r*W + c].z = img(c,r, 0);
            arr[r*W + c].w = 0;
        }
    }
    
    
    

    //device storage
    float4 *d_in = 0;
    uchar4 *d_out = 0;
     
    hipMalloc(&d_in, W*H*sizeof(float4));
    hipMemcpy(d_in, arr, W*H*sizeof(float4), hipMemcpyHostToDevice);
    
    hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
    hipGraphicsResourceGetMappedPointer((void **)&d_out, NULL,cuda_pbo_resource);    
    
    //kernelLauncher     
    const dim3 blockSize(TX, TY);
    const dim3 gridSize = dim3((W + TX - 1)/TX, (H + TY - 1)/TY);
    distanceKernel<<<gridSize, blockSize>>>(d_out,d_in, W, H, loc, g_fAnim);  
    hipDeviceSynchronize();
    hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0); 
}


// Display model
static void display(){
    
    cudarun();
    glClearColor(0.0,0.34,0.46,1.0);
    glClearDepth(1.0);
    glClear(GL_COLOR_BUFFER_BIT|GL_DEPTH_BUFFER_BIT);
    glEnable(GL_LIGHTING);
    glEnable(GL_DEPTH_TEST);
    glEnable(GL_COLOR_MATERIAL);
    glEnable(GL_LIGHT0);
    glEnable(GL_LIGHT1);
    glShadeModel(GL_SMOOTH);
    glEnable(GL_CULL_FACE);
    glCullFace(GL_BACK);
    glEnable(GL_COLOR_MATERIAL);
    glEnable(GL_NORMALIZE);
    
    
    glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, W, H, 0, GL_RGBA,GL_UNSIGNED_BYTE, NULL);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);    
    glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    
    //Textue map
    glEnable(GL_TEXTURE_2D);
    glActiveTexture(GL_TEXTURE_2D);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glBegin(GL_QUADS);
    	glTexCoord2f(0.0f, 0.0f); glVertex2f(0,0);
    	glTexCoord2f(0.0f, 1.0f); glVertex2f(0,H);
    	glTexCoord2f(1.0f, 1.0f); glVertex2f(W,H);
    	glTexCoord2f(1.0f, 0.0f); glVertex2f(W,0);
    glEnd();
    glPopMatrix();
    

    glFlush();
    glDepthFunc(GL_LEQUAL); 
    glHint(GL_PERSPECTIVE_CORRECTION_HINT, GL_NICEST);
    glutSwapBuffers();  
    g_fAnim += 0.05f;
    glDisable(GL_TEXTURE_2D);
    
}


// Reshape window
static void reshape(int w, int h){
    glViewport(0, 0, (GLsizei) w, (GLsizei) h);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(0, W, H, 0);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
}

// Pixel Buffer generate
static void initPixelBuffer() {
    glGenBuffers(1, &pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, 4*W*H*sizeof(GLubyte), 0, GL_STREAM_DRAW);

    glGenTextures(1, &tex);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo,cudaGraphicsMapFlagsWriteDiscard);
}


// Handler for animation
static void animate(void){
     angle1  = 0.01f*glutGet(GLUT_ELAPSED_TIME);
     angle2 =  0.01f*glutGet(GLUT_ELAPSED_TIME);
     glutPostRedisplay();

}

// Handler for timer event
static void timerEvent(int value){
    if (glutGetWindow()) {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

// Free Buffer and Texture
static void exitfunc() {
    if (pbo) {
        hipGraphicsUnregisterResource(cuda_pbo_resource);
        glDeleteBuffers(1, &pbo);
        glDeleteTextures(1, &tex);
    }
}

// Handler for mous event
static void mouskey(int button,int state, int x, int y){
    if (state == GLUT_DOWN)    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)    {
        mouse_buttons = 0;
    }
    mouse_old_x = x;
    mouse_old_y = y;
        
 }


// Handler for key event
static void keyboard(unsigned char key, int x, int y) {
  if(x==0||y==0) return;
    switch (key){
        case (27) :
            if (key==27||key=='q'||key=='Q')
                exit(EXIT_SUCCESS);
            else
                glutDestroyWindow(glutGetWindow());
                return;        
    }
}


static void motion(int x, int y){
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);
    if (mouse_buttons & 1){
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4){
        translate_z += dy * 0.01f;
    }
    mouse_old_x = x;
    mouse_old_y = y;
}


int main(int argc, char** argv) {
    hipDeviceProp_t  prop;
    hipGetDeviceProperties(&prop, 0);
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE|GLUT_DEPTH);
    glutInitWindowSize(W, H);
    glutInitWindowPosition(20, 20);
    glutCreateWindow("Heat Flux");     
    glutDisplayFunc(display);
    glutReshapeFunc(reshape);
    initPixelBuffer();
    glutIdleFunc(animate);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    glutMouseFunc(mouskey);
    glutMotionFunc(motion);
    glutKeyboardFunc(keyboard);
    glutMainLoop();
    atexit(exitfunc);
    return 0;
}
